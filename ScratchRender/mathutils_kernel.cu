
#include <math.h>
#include <iostream>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>

#include "./mathutils_kernel.cuh"

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            std::cerr <<"CUDA Error:\nFile " << __FILE__ << "\nLine " << __LINE__ << "\nError " << hipGetErrorString(err);    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            std::cerr <<"CUBLAS Error:\nFile " << __FILE__ << "\nLine " << __LINE__ << "\nCode " << status; \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)



void mult_mm(float* A, float* B, float* C, int N) {

    hipError_t error;
    hipblasStatus_t state;
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    // Initialize device pointers.
    float* d_A, * d_B, * d_C;

    // Allocate device memory.
    cudacall(hipMalloc((void**)&d_A, N * sizeof(float)));
    cudacall(hipMalloc((void**)&d_B, N * sizeof(float)));
    cudacall(hipMalloc((void**)&d_C, N * sizeof(float)));

    // Transfer arrays a and b to device.
    cudacall(hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice));
    cudacall(hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice));

    // *********** multiplication A*B

    int m = N, n = N, k = N;
    int lda = N, ldb = N, ldc = N;
    const float alf = 1;
    const float bet = 0;
    const float* alpha = &alf;
    const float* beta = &bet;

    cublascall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_B, ldb, d_A, lda, beta, d_C, ldc));

    // ***********

    cudacall(hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    cudacall(hipFree(d_A));
    cudacall(hipFree(d_B));
    cudacall(hipFree(d_C));

    cublascall(hipblasDestroy(handle));
    //hipDeviceReset();
}

void mult_mv(float* A, float* B, float* C, int N) {

    hipError_t error;
    hipblasStatus_t state;
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    float* d_A, * d_B, * d_C;

    // Allocate device memory.
    cudacall(hipMalloc((void**)&d_A, N * N * sizeof(float)));
    cudacall(hipMalloc((void**)&d_B, N * sizeof(float)));
    cudacall(hipMalloc((void**)&d_C, N * sizeof(float)));

    // Transfer arrays a and b to device.
    cudacall(hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice));
    cudacall(hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice));

    // *********** multiplication A*v
    int m = N, n = N;
    int lda = N;
    const float alf = 1;
    const float bet = 0;
    const float* alpha = &alf;
    const float* beta = &bet;

    cublascall(hipblasSgemv(handle, HIPBLAS_OP_T, m, n, alpha, d_A, lda, d_B, 1, beta, d_C, 1));

    // ***********

    cudacall(hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    cudacall(hipFree(d_A));
    cudacall(hipFree(d_B));
    cudacall(hipFree(d_C));

    cublascall(hipblasDestroy(handle));
    //hipDeviceReset();
}

void invert(float** src, float** dst, int n, int batchSize)
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int* P, * INFO;

    cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
    cudacall(hipMalloc(&INFO, batchSize * sizeof(int)));

    int lda = n;

    float** A = (float**)malloc(batchSize * sizeof(float*));
    float** A_d, * A_dflat;

    cudacall(hipMalloc(&A_d, batchSize * sizeof(float*)));
    cudacall(hipMalloc(&A_dflat, n * n * batchSize * sizeof(float)));

    A[0] = A_dflat;
    for (int i = 1; i < batchSize; i++)
        A[i] = A[i - 1] + (n * n);

    cudacall(hipMemcpy(A_d, A, batchSize * sizeof(float*), hipMemcpyHostToDevice));

    for (int i = 0; i < batchSize; i++)
        cudacall(hipMemcpy(A_dflat + (i * n * n), src[i], n * n * sizeof(float), hipMemcpyHostToDevice));


    cublascall(hipblasSgetrfBatched(handle, n, A_d, lda, P, INFO, batchSize));

    int INFOh[1];
    cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < batchSize; i++)
        if (INFOh[i] != 0)
        {
            fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
            cudacall(hipDeviceReset());
            exit(EXIT_FAILURE);
        }

    float** C = (float**)malloc(batchSize * sizeof(float*));
    float** C_d, * C_dflat;

    cudacall(hipMalloc(&C_d, batchSize * sizeof(float*)));
    cudacall(hipMalloc(&C_dflat, n * n * batchSize * sizeof(float)));
    C[0] = C_dflat;
    for (int i = 1; i < batchSize; i++)
        C[i] = C[i - 1] + (n * n);
    cudacall(hipMemcpy(C_d, C, batchSize * sizeof(float*), hipMemcpyHostToDevice));
    cublascall(hipblasSgetriBatched(handle, n, (const float**)A_d, lda, P, C_d, lda, INFO, batchSize));

    cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));

    for (int i = 0; i < batchSize; i++)
        if (INFOh[i] != 0)
        {
            fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
            hipDeviceReset();
            exit(EXIT_FAILURE);
        }
    for (int i = 0; i < batchSize; i++)
        cudacall(hipMemcpy(dst[i], C_dflat + (i * n * n), n * n * sizeof(float), hipMemcpyDeviceToHost));

    cudacall(hipFree(A_d)); cudacall(hipFree(A_dflat)); free(A);
    cudacall(hipFree(C_d)); cudacall(hipFree(C_dflat)); free(C);
    cudacall(hipFree(P)); cudacall(hipFree(INFO));
    
    cublascall(hipblasDestroy(handle));
}